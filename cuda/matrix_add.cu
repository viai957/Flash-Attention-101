#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "cuda_common.cuh"

typedef float EL_TYPE;

__global__ void cuda_matrix_add(EL_TYPE *OUT, EL_TYPE *A, EL_TYPE *B, int NUM_ROWS, int NUM_COLS)
{
    int row_index = blockIdx.y * blockDim.y + threadIdx.y;
    int col_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (row_index < NUM_ROWS && col_index < NUM_COLS)
    {
        size_t index = static_cast<size_t>(row_index) * NUM_COLS + col_index; // A[row_index][col_index]
        OUT[index] = A[index] + B[index];
    }
}

void test_matrix_add(int NUM_ROWS, int NUM_COLS, int ROWS_block_size, int COLS_block_size) 
{
    EL_TYPE *A, *B, *OUT;
    EL_TYPE *d_A, *d_B, *d_OUT;

    // Allocate the matrices on the host device
    A = (EL_TYPE *)malloc(sizeof(EL_TYPE) * NUM_ROWS * NUM_COLS);
    B = (EL_TYPE *)malloc(sizeof(EL_TYPE) * NUM_ROWS * NUM_COLS);
    OUT = (EL_TYPE *)malloc(sizeof(EL_TYPE) * NUM_ROWS * NUM_COLS);

    // Initialize the matrices with random values
    for (int i = 0; i < NUM_ROWS; i++)
    {
        for (int j = 0; j < NUM_COLS; j++)
        {
            size_t index = static_cast<size_t>(i) * NUM_COLS + j;
            A[index] = rand() % 100;
            B[index] = rand() % 100;
        }
    }

    // Allocate device memory for a
    CUDA_CHECK(hipMalloc((void **)&d_A, sizeof(EL_TYPE) * NUM_ROWS * NUM_COLS));
    CUDA_CHECK(hipMalloc((void **)&d_B, sizeof(EL_TYPE) * NUM_ROWS * NUM_COLS));
    CUDA_CHECK(hipMalloc((void **)&d_OUT, sizeof(EL_TYPE) * NUM_ROWS * NUM_COLS));

    // Transfer the matrices to the device
    CUDA_CHECK(hipMemcpy(d_A, A, sizeof(EL_TYPE) * NUM_ROWS * NUM_COLS, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, sizeof(EL_TYPE) * NUM_ROWS * NUM_COLS, hipMemcpyHostToDevice));

    hipEvent_t start_kernel, stop_kernel;
    CUDA_CHECK(hipEventCreate(&start_kernel));
    CUDA_CHECK(hipEventCreate(&stop_kernel));

    CUDA_CHECK(hipEventRecord(start_kernel));
    
    // Define the launch grid
    int num_blocks_ROWS = (NUM_ROWS + ROWS_block_size - 1) / ROWS_block_size; // ceil(NUM_ROWS / ROWS_block_size)
    int num_blocks_COLS = (NUM_COLS + COLS_block_size - 1) / COLS_block_size; // ceil(NUM_COLS / COLS_block_size)
    printf("Matrix Add - M: %d, N: %d will be processed by (%d x %d) blocks of size (%d x %d)\n", NUM_ROWS, NUM_COLS, num_blocks_ROWS, num_blocks_COLS, ROWS_block_size, COLS_block_size);
    dim3 grid(num_blocks_COLS, num_blocks_ROWS, 1);
    dim3 block(COLS_block_size, ROWS_block_size, 1);
    // Run the kernel
    cuda_matrix_add<<<grid, block>>>(d_OUT, d_A, d_B, NUM_ROWS, NUM_COLS);

    // Check for launch errors
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipEventRecord(stop_kernel));
    CUDA_CHECK(hipEventSynchronize(stop_kernel));

    // Calculate elapsed milliseconds
    float milliseconds_kernel = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds_kernel, start_kernel, stop_kernel));
    printf("Matrix Add - Elapsed time: %f ms\n", milliseconds_kernel);

    // Copy back the result from the device to the host
    CUDA_CHECK(hipMemcpy(OUT, d_OUT, sizeof(EL_TYPE) * NUM_ROWS * NUM_COLS, hipMemcpyDeviceToHost));

    // Free the memory on the device
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_OUT));

    // Time the operation 
    struct timeval start_check, end_check;
    gettimeofday(&start_check, NULL);

    for (int i = 0; i < NUM_ROWS; i++)
    {
        for (int j = 0; j < NUM_COLS; j++)
        {
            size_t index = static_cast<size_t>(i) * NUM_COLS + j;
            if (OUT[index] != A[index] + B[index])
            {
                printf("Error at index (%d, %d): %.2f != %.2f + %.2f\n", i, j, OUT[index], A[index], B[index]);
                exit(1);
            }
        }
    }

    // Calculate elapsed time
    gettimeofday(&end_check, NULL);
    float elapsed = (end_check.tv_sec - start_check.tv_sec) * 1000.0 + (end_check.tv_usec - start_check.tv_usec) / 1000.0;
    printf("Matrix Add - Check elapsed time: %f ms\n", elapsed);

    printf("Matrix Add - Result OK\n");

    // Free the memory on the host
    free(A);
    free(B);
    free(OUT);
}

int main()
{
    // set your seed
    srand(0);

    test_matrix_add(10000, 10000, 16, 16);

}